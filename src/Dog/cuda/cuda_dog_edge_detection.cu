#include "hip/hip_runtime.h"
//
// Created by Palnit on 2024. 01. 21.
//

#include "include/Dog/cuda/cuda_dog_edge_detection.cuh"
#include "include/general/cuda/gauss_blur.cuh"

__global__ void DifferenceOfGaussian(float* kernel1,
                                     float* kernel2,
                                     float* finalKernel,
                                     int kernelSize) {
    uint32_t x = threadIdx.x;
    uint32_t y = threadIdx.y;

    *(finalKernel + x + (y * kernelSize)) =
        *(kernel1 + x + (y * kernelSize)) - *(kernel2 + x + (y * kernelSize));
}

void CudaDogDetector::DogDetect() {
    float* dest1;
    float* dest2;

    float* kernel1;
    float* kernel2;
    float* finalKernel;

    dim3 threads(32, 32);
    dim3 block
        (m_w / threads.x + (m_w % threads.x == 0 ? 0 : 1),
         m_h / threads.y
             + (m_h % threads.y == 0 ? 0 : 1));

    dim3 block2
        ((m_w / (threads.x - m_gaussKernelSize + 1))
             + (m_w % (threads.x - m_gaussKernelSize + 1) == 0 ? 0 : 1),
         (m_h / (threads.y - m_gaussKernelSize + 1))
             + (m_h % (threads.y - m_gaussKernelSize + 1) == 0 ? 0 : 1));

    hipMalloc((void**) &kernel1,
               sizeof(float) * m_gaussKernelSize * m_gaussKernelSize);
    hipMalloc((void**) &kernel2,
               sizeof(float) * m_gaussKernelSize * m_gaussKernelSize);
    hipMalloc((void**) &finalKernel,
               sizeof(float) * m_gaussKernelSize * m_gaussKernelSize);
    hipMalloc((void**) &dest1, sizeof(float) * m_w * m_h);
    hipMalloc((void**) &dest2, sizeof(float) * m_w * m_h);

    dim3 gauss(m_gaussKernelSize, m_gaussKernelSize);
    hipEventRecord(m_timers.All_start);
    hipEventRecord(m_timers.GrayScale_start);
    convertToGreyScale<<<block, threads>>>(m_src, dest1, m_w, m_h);
    hipEventRecord(m_timers.GrayScale_stop);
    hipEventSynchronize(m_timers.GrayScale_stop);

    hipEventRecord(m_timers.Gauss1Creation_start);
    GetGaussian<<<1, gauss>>>(kernel1, m_gaussKernelSize, m_standardDeviation1);
    hipEventRecord(m_timers.Gauss1Creation_stop);
    hipEventSynchronize(m_timers.Gauss1Creation_stop);

    hipEventRecord(m_timers.Gauss2Creation_start);
    GetGaussian<<<1, gauss>>>(kernel2, m_gaussKernelSize, m_standardDeviation2);
    hipEventRecord(m_timers.Gauss2Creation_stop);
    hipEventSynchronize(m_timers.Gauss2Creation_stop);

    hipEventRecord(m_timers.DifferenceOfGaussian_start);
    DifferenceOfGaussian<<<1, gauss>>>(kernel1,
                                       kernel2,
                                       finalKernel,
                                       m_gaussKernelSize);
    hipEventRecord(m_timers.DifferenceOfGaussian_stop);
    hipEventSynchronize(m_timers.DifferenceOfGaussian_stop);

    hipEventRecord(m_timers.Convolution_start);
    GaussianFilter<<<block2, threads>>>(dest1,
                                        dest2,
                                        finalKernel,
                                        m_gaussKernelSize,
                                        m_w,
                                        m_h);
    hipEventRecord(m_timers.Convolution_stop);
    hipEventSynchronize(m_timers.Convolution_stop);

    CopyBack<<<block, threads>>>(m_src, dest2, m_w, m_h);
    hipEventRecord(m_timers.All_stop);
    hipEventSynchronize(m_timers.All_stop);

    hipEventElapsedTime(&m_timings.All_ms,
                         m_timers.All_start,
                         m_timers.All_stop);
    hipEventElapsedTime(&m_timings.GrayScale_ms,
                         m_timers.GrayScale_start,
                         m_timers.GrayScale_stop);
    hipEventElapsedTime(&m_timings.Gauss1Creation_ms,
                         m_timers.Gauss1Creation_start,
                         m_timers.Gauss1Creation_stop);
    hipEventElapsedTime(&m_timings.Gauss2Creation_ms,
                         m_timers.Gauss2Creation_start,
                         m_timers.Gauss2Creation_stop);
    hipEventElapsedTime(&m_timings.DifferenceOfGaussian_ms,
                         m_timers.DifferenceOfGaussian_start,
                         m_timers.DifferenceOfGaussian_stop);
    hipEventElapsedTime(&m_timings.Convolution_ms,
                         m_timers.Convolution_start,
                         m_timers.Convolution_stop);

    hipFree(dest1);
    hipFree(dest2);
    hipFree(kernel1);
    hipFree(kernel2);
    hipFree(finalKernel);
}
